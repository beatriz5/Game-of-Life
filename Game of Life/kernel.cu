#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <windows.h>
#include <conio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
using namespace std;

//Kernel, deals with movements. Nvidia GPU Global Memory
__global__ void gameKernel(int* Md, int* Nd, int* Pd, int Width, int h)
{
	int currently = threadIdx.x + blockDim.x * blockIdx.x;
	int Row = currently / Width;
	int Col = currently - (Width * Row);

	int Pvalue = 0;
	if (Col > 0) { if (1 == Md[currently - 1]) { Pvalue += 1; } }//Left
	if (Col < Width - 1) { if (1 == Md[currently + 1]) { Pvalue += 1; } }//Right
	if (Row > 0) { if (1 == Md[currently - Width]) { Pvalue += 1; } }//Up
	if (Row < Width - 1) { if (1 == Md[currently + Width]) { Pvalue += 1; } }//Down
	if (Col > 0 && Row > 0) { if (1 == Md[currently - Width - 1]) { Pvalue += 1; } }//Left diagonal up
	if (Col < Width - 1 && Row > 0) { if (1 == Md[currently - Width + 1]) { Pvalue += 1; } }//Right diagonal up
	if (Col > 0 && Row < Width - 1) { if (1 == Md[currently + Width - 1]) { Pvalue += 1; } }//Left diagonal down
	if (Col < Width - 1 && Row < Width - 1) { if (1 == Md[currently + Width + 1]) { Pvalue += 1; } }//Right diagonal down

	int value = 0;
	if (0 == Md[Row * Width + Col]) { if (Pvalue == 3) { value = 1; } }

	if (1 == Md[Row * Width + Col]) {
		if ((Pvalue == 2 || Pvalue == 3)) { value = 1; }
		else { value = 0; }
	}

	Nd[currently] = value;
	Pd[currently] = value;
}


//Function that shows the board
void printBoard(int* board, int width, int height) {
	printf("\n\n-----------------------------------------------------------------------------------------------------------------\n\n");
	char columnLetter = 'A';
	cout << "    " << columnLetter << " ";
	for (int i = 1; i < width; i++) {
		columnLetter++;
		cout << columnLetter << " ";
		if (columnLetter == 'Z') { columnLetter = '@'; }
	}

	cout << "\n\n";
	int rowNumber = 0;
	for (int j = 0; j < height; j++) {
		if (rowNumber < 9)
			cout << " " << rowNumber + 1 << "  ";
		else
			cout << " " << rowNumber + 1 << " ";
		for (int k = 0; k < width; k++) {
			if (board[(j * width) + k] == 1) {
				cout << "X" << " ";
			}
			else {
				cout << " " << " ";
			}
		}
		printf("\n");
		rowNumber++;
	}
}

//Function that launches the kernel
void playOnDevice(int* M, int* N, int* P, int w, const int h)
{
	int size = w * h * sizeof(float);
	int* Md;
	int* Pd;
	int* Nd;

	hipMalloc(&Md, size); //allocate the memory on the GPU
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice); //Host to device
	hipMalloc(&Nd, size);				hipMalloc(&Pd, size); //allocate the memory on the GPU

	gameKernel << < 1, w* h >> > (Md, Nd, Pd, w, h); //launch kernel

	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);			hipMemcpy(N, Nd, size, hipMemcpyDeviceToHost);
	hipFree(Md); hipFree(Nd);  hipFree(Pd);
}


//To ask for the width and validate it
int* askWidth(int threadsBlockX) {
	int* width = new int;
	cout << "\nEnter the width of the board: ";
	cin >> *width;
	while (cin.fail() || (*width < 0) || (*width > threadsBlockX)) {
		cout << "\nERROR: incorrect width, please try again: ";
		cin.clear();
		cin.ignore(256, '\n');
		cin >> *width;
	}
	return width;
}

//To ask for the height and validate it
int* askHeight(int threadsBlockY) {
	int* height = new int;
	cout << "\nEnter the height of the board: ";
	cin >> *height;
	while (cin.fail() || (*height < 0) || (*height > threadsBlockY)) {
		cout << "\nERROR: incorrect height, please try again: ";
		cin.clear();
		cin.ignore(256, '\n');
		cin >> *height;
	}
	return height;
}


//Function that creates the board with random 1's and 0's
int* createBoard(int height, int width) {

	time_t t;
	srand((unsigned)time(&t));
	int boardSize = width * height;
	int* board = new int[boardSize];
	for (int i = 0; i < height * width; i++) {
		board[i] = (int)rand() % 2;
	}
	return board;
}


//******************
//** MAIN FUNCION **
//******************
int main() {
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	int threadsPerBlock = properties.maxThreadsPerBlock;
	int threadsBlockX = properties.maxThreadsDim[0];
	int threadsBlockY = properties.maxThreadsDim[1];
	int* board = new int;
	int* height = new int;
	int* width = new int;
	int* boardR = new int;
	cout << "\nWelcome to the Game of Life \n";
	boolean m = true;
	while (m) {
		width = askWidth(threadsBlockX);
		height = askHeight(threadsBlockY);
		if (((*width) * (*height)) <= threadsPerBlock) {
			m = false;
		}
		else { cout << "\nToo many threads, decrease dimensions\n"; }
	}

	board = createBoard(*width, *height);
	boardR = createBoard(*width, *height);

	char mode;
	cout << "\nChoose the game mode (a/m): ";
	cin >> mode;
	while (cin.fail() || ((mode != 'a') && (mode != 'm'))) {
		cout << "\nERROR: wrong mode, please try again: ";
		cin.clear();
		cin.ignore(256, '\n');
		cin >> mode;
	}

	if (mode == 'a') {
		printBoard(board, *width, *height);
		while (!kbhit())
		{
			playOnDevice(board, board, boardR, *width, *height);
			printBoard(boardR, *width, *height);
			Sleep(500);
		}
	}
	else {
		printBoard(board, *width, *height);
		while (mode != 's')
		{
			playOnDevice(board, board, boardR, *width, *height);
			cout << "\nTo continue type any letter, and to quit 's': ";
			cin >> mode;
			if (mode != 's') { printBoard(boardR, *width, *height);}
		}
	}

	system("pause");
	return(0);
}

